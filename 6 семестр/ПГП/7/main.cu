#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <thrust/gather.h>

#define THREADS_PER_BLOCK 32

__global__ void dot(int *a, int *b, int *c)
{
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    temp[threadIdx.x] = a[index] * b[index];

    __syncthreads();

    if (threadIdx.x == 0)
    {
        int sum = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
        {
            sum += temp[i];
        }
        atomicAdd(c, sum);
    }
}

__global__ void transpose(int *in, int *out, int N)
{
    __shared__ int tile[THREADS_PER_BLOCK][THREADS_PER_BLOCK + 1];
    int x = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    int y = blockIdx.y * 512 + threadIdx.y;

    for (int j = 0; j < THREADS_PER_BLOCK; j += blockDim.y)
    {
        if (x < N && y + j < N)
        {
            int index_in = (y + j) * N + x;
            tile[threadIdx.y + j][threadIdx.x] = in[index_in];
        }
    }

    __syncthreads();

    x = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;
    y = blockIdx.x * THREADS_PER_BLOCK + threadIdx.y;
    for (int j = 0; j < THREADS_PER_BLOCK; j += blockDim.y)
    {
        if (x < N && y + j < N)
        {
            int index_out = (y + j) * N + x;
            out[index_out] = tile[threadIdx.x][threadIdx.y + j];
        }
    }
}

__host__ std::vector<float> compare_dot_product(int N)
{
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    int size = N * sizeof(int);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    std::vector<float> statistics{(float)N};

    // allocate space for the variables on the device
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, sizeof(int));

    // allocate space for the variables on the host
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(sizeof(int));

    // this is our ground truth
    // int sumTest = N;
    // generate numbers
    for (int i = 0; i < N; i++)
    {
        a[i] = 1;
        b[i] = 1;
    }

    *c = 0;

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(start);

    dot<<<(N / THREADS_PER_BLOCK - 1) + THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    statistics.push_back(milliseconds);

    // printf("CUDA API\n");
    // printf("Time: %f\n", milliseconds);
    // printf("Are sums equal: %d\n\n", sumTest == *c);

    thrust::device_vector<int> a_thrust(N, 1), b_thrust(N, 1);

    for (int i = 0; i < N; i++)
    {
        a_thrust[i] = 1;
        b_thrust[i] = 1;
    }

    hipEventRecord(start);

    float result = thrust::inner_product(a_thrust.begin(), a_thrust.end(), b_thrust.begin(), 0.0f);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    statistics.push_back(milliseconds);

    // printf("THRUST \n");
    // printf("Time: %f\n", milliseconds);
    // printf("Are sums equal: %d\n", sumTest == result);

    free(a);
    free(b);
    free(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    // hipFree(c);

    return statistics;
}

__host__ std::vector<float> compare_transpose(int N)
{

    int *a, *b;
    int *dev_a, *dev_b;
    int size = N * N * sizeof(int);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    std::vector<float> statistics{(float)N};

    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);

    a = (int *)malloc(size);

    for (int i = 0; i < N * N; i++)
    {
        a[i] = i;
    }
    b = (int *)malloc(size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

    hipEventRecord(start);

    transpose<<<(N / THREADS_PER_BLOCK - 1) + 512, THREADS_PER_BLOCK>>>(dev_a, dev_b, N);

    hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    statistics.push_back(milliseconds);

    thrust::host_vector<int> h_A(N * N);
    thrust::host_vector<int> h_AT(N * N);
    thrust::device_vector<int> d_A(N * N);
    thrust::device_vector<int> d_AT(N * N);

    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = i;
    }

    int map[N * N];
    for (int i = 0; i < N * N; i++)
    {
        map[i] = (i % N) * N + (i / N);
    }
    thrust::device_vector<int> d_map(map, map + N * N);

    hipEventRecord(start);

    thrust::gather(d_map.begin(), d_map.end(), d_A.begin(), d_AT.begin());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    statistics.push_back(milliseconds);

    // printf("THRUST \n");
    // printf("Time: %f\n", milliseconds);
    // printf("Are sums equal: %d\n", sumTest == result);

    free(a);
    free(b);

    hipFree(dev_a);
    hipFree(dev_b);
    // hipFree(c);

    return statistics;
}

template <typename T>
void write_to_file(std::vector<std::vector<T>> data, std::string filename)
{
    std::ofstream out(filename);
    for (auto row : data)
    {
        std::string row_out = "";
        for (auto item : row)
        {
            row_out += std::to_string(item) + ";";
        }
        row_out.pop_back();
        out << row_out << std::endl;
    }
}

int main()
{
    std::vector<std::vector<float>> stats;
    for (int i = 1; i < 1024 * 1024; i *= 2)
    {
        stats.push_back(compare_dot_product(i));
    }
    write_to_file(stats, "dot_product.csv");
    stats.clear();
    for (int i = 2; i <= 1024; i *= 2)
    {
        stats.push_back(compare_transpose(i));
    }
    write_to_file(stats, "transpose.csv");

    return 0;
}