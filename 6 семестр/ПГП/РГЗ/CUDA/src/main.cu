#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <chrono>
#include <thread>
#include <fstream>
#include "utility.h"
#include "cuda_handler.h"
#include "matrix_handler.h"

using namespace std;

int main()
{
    float *input_matrix_1, *input_matrix_2, *output_matrix;
    input_matrix_1 = (float *)malloc(NUM_ELEMENTS * sizeof(float));
    input_matrix_2 = (float *)malloc(NUM_ELEMENTS * sizeof(float));
    output_matrix = (float *)malloc(NUM_ELEMENTS * sizeof(float));

    init_matrix(input_matrix_1);
    this_thread::sleep_for(chrono::seconds(1));
    init_matrix(input_matrix_2);
    init_zero_matrix(output_matrix);

    if (PRINT_MODE)
    {
        cout << "BASE MATRICES:" << endl;
        cout << "input_matrix_1:" << endl;
        print_matrix(input_matrix_1);
        cout << "input_matrix_2:" << endl;
        print_matrix(input_matrix_2);
        cout << "output_matrix:" << endl;
        print_matrix(output_matrix);
    }

    float *device_input_matrix_1, *device_input_matrix_2, *device_output_matrix;
    hipMalloc(&device_input_matrix_1, NUM_ELEMENTS * sizeof(float));
    hipMalloc(&device_input_matrix_2, NUM_ELEMENTS * sizeof(float));
    hipMalloc(&device_output_matrix, NUM_ELEMENTS * sizeof(float));
    hipMemcpy(device_input_matrix_1, input_matrix_1, NUM_ELEMENTS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_input_matrix_2, input_matrix_2, NUM_ELEMENTS * sizeof(float), hipMemcpyHostToDevice);

    cout << "KERNEL TRANSPOSING:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_kernel_transpose = 0.0f;
    elapsed_time_kernel_transpose = kernel_transpose(device_input_matrix_1, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    transpose_check(input_matrix_1, output_matrix);
    cout << "Time: " << elapsed_time_kernel_transpose << " ms" << endl;
    cout << endl;

    cout << "THRUST TRANSPOSING:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_thrust_transpose = 0.0f;
    elapsed_time_thrust_transpose = thrust_transpose(device_input_matrix_1, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    transpose_check(input_matrix_1, output_matrix);
    cout << "Time: " << elapsed_time_thrust_transpose << " ms" << endl;
    cout << endl;

    cout << "CUBLAS TRANSPOSING:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_cublas_transpose = 0.0f;
    elapsed_time_cublas_transpose = cublas_transpose(device_input_matrix_1, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    transpose_check(input_matrix_1, output_matrix);
    cout << "Time: " << elapsed_time_cublas_transpose << " ms" << endl;
    cout << endl;

    cout << "KERNEL MATRIX MULTIPLICATION:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_kernel_matrix_multiplication = 0.0f;
    elapsed_time_kernel_matrix_multiplication = kernel_matrix_multiply(device_input_matrix_1, device_input_matrix_2, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    multiply_check(input_matrix_1, input_matrix_2, output_matrix);
    cout << "Time: " << elapsed_time_kernel_matrix_multiplication << " ms" << endl;
    cout << endl;

    cout << "THRUST MATRIX MULTIPLICATION:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_thrust_matrix_multiplication = 0.0f;
    elapsed_time_thrust_matrix_multiplication = thrust_matrix_multiply(device_input_matrix_1, device_input_matrix_2, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    multiply_check(input_matrix_1, input_matrix_2, output_matrix);
    cout << "Time: " << elapsed_time_thrust_matrix_multiplication << " ms" << endl;
    cout << endl;

    cout << "CUBLAS MATRIX MULTIPLICATION:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_cublas_matrix_multiplication = 0.0f;
    elapsed_time_cublas_matrix_multiplication = cublas_matrix_multiply(device_input_matrix_1, device_input_matrix_2, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    multiply_check(input_matrix_1, input_matrix_2, output_matrix);
    cout << "Time: " << elapsed_time_cublas_matrix_multiplication << " ms" << endl;
    cout << endl;

    cout << "KERNEL SCALAR MULTIPLICATION:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_kernel_scalar_multiplication = 0.0f;
    elapsed_time_kernel_scalar_multiplication = kernel_scalar_multiply(device_input_matrix_1, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    scalar_check(input_matrix_1, output_matrix);
    cout << "Time: " << elapsed_time_kernel_scalar_multiplication << " ms" << endl;
    cout << endl;

    cout << "THRUST SCALAR MULTIPLICATION:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_thrust_scalar_multiplication = 0.0f;
    elapsed_time_thrust_scalar_multiplication = thrust_scalar_multiply(device_input_matrix_1, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    scalar_check(input_matrix_1, output_matrix);
    cout << "Time: " << elapsed_time_thrust_scalar_multiplication << " ms" << endl;
    cout << endl;

    cout << "CUBLAS SCALAR MULTIPLICATION:" << endl;
    init_zero_matrix(output_matrix);
    float elapsed_time_cublas_scalar_multiplication = 0.0f;
    elapsed_time_cublas_scalar_multiplication = cublas_scalar_multiply(device_input_matrix_1, device_output_matrix);
    hipMemcpy(output_matrix, device_output_matrix, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);
    scalar_check(input_matrix_1, output_matrix);
    cout << "Time: " << elapsed_time_cublas_scalar_multiplication << " ms" << endl;
    cout << endl;

    cout << "TOTAL RESULTS:" << endl;
    cout << "MATRIX SIZE = " << MATRIX_SIZE << " x " << MATRIX_SIZE << " (" << NUM_ELEMENTS << " elements)" << endl;
    cout << setw(40) << "raw CUDA, ms" << setw(15) << "Thrust, ms"
         << setw(15) << "cuBLSAS, ms" << endl;
    cout << fixed;
    cout << setw(25) << "Matrix transposing" << setw(15) << elapsed_time_kernel_transpose << setw(15) << elapsed_time_thrust_transpose << setw(15) << elapsed_time_cublas_transpose << endl;
    cout << setw(25) << "Matrix multiplication" << setw(15) << elapsed_time_kernel_matrix_multiplication << setw(15) << elapsed_time_thrust_matrix_multiplication << setw(15) << elapsed_time_cublas_matrix_multiplication << endl;
    cout << setw(25) << "Scalar multiplication" << setw(15) << elapsed_time_kernel_scalar_multiplication << setw(15) << elapsed_time_thrust_scalar_multiplication << setw(15) << elapsed_time_cublas_scalar_multiplication << endl;

    if (FILE_OUTPUT)
    {
        ofstream file;
        file.open("results.csv", ios::app);

        if (file.tellp() == streampos(0))
        {
            file << "Matrix size (n^2); MT (CUDA); MT (Thrust); MT (cuBLAS); MM (CUDA); MM (Thrust); MM (cuBLAS); SM (CUDA); SM (Thrust); SM (cuBLAS);" << endl;
        }

        file << MATRIX_SIZE << ";"
             << elapsed_time_kernel_transpose << ";" << elapsed_time_thrust_transpose << ";" << elapsed_time_cublas_transpose << ";"
             << elapsed_time_kernel_matrix_multiplication << ";" << elapsed_time_thrust_matrix_multiplication << ";" << elapsed_time_cublas_matrix_multiplication << ";"
             << elapsed_time_kernel_scalar_multiplication << ";" << elapsed_time_thrust_scalar_multiplication << ";" << elapsed_time_cublas_scalar_multiplication << ";"
             << endl;

        file.close();
    }

    return EXIT_SUCCESS;
}