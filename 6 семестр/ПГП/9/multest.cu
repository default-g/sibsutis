#include <math.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void scalar_multiply(int *a, int *b, int *c, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int start = tid * size;
    int end = start + size;
    for (int i = start; i < end && i < size; i++)
    {
        c[tid * size + i] = a[tid * size + i] * b[tid * size + i];
    }
}

int main(int argc, char **argv)
{
    FILE *csv_file = fopen("results_9/3.csv", "a+");
    int size = pow(2, 17);
    int chunk_size = atoi(argv[1]);
    const int num_threads = 1024;
    int num_blocks = (size + num_threads - 1) / num_threads;
    int num_chunks = (size + chunk_size - 1) / chunk_size;
    int *h_a = new int[size];
    int *h_b = new int[size];
    int *h_c = new int[size];
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));
    srand(time(nullptr));
    for (int i = 0; i < size; i++)
    {
        h_a[i] = rand();
        h_b[i] = rand();
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    for (int i = 0; i < num_chunks; i += 2)
    {
        int offset = num_chunks / 2;

        hipMemcpyAsync(&d_a[i], &h_a[i], chunk_size * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(&d_a[i + offset], &h_a[i + offset], chunk_size * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(&d_b[i], &h_b[i], chunk_size * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(&d_b[i + offset], &h_b[i + offset], chunk_size * sizeof(int), hipMemcpyHostToDevice, stream1);
        scalar_multiply<<<num_blocks, num_threads, 0, stream0>>>(&d_a[i], &d_b[i], &d_c[i], chunk_size);
        scalar_multiply<<<num_blocks, num_threads, 0, stream1>>>(&d_a[i + offset], &d_b[i + offset], &d_c[i + offset], chunk_size);
        hipMemcpyAsync(&h_c[i], &d_c[i], chunk_size * sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(&h_c[i + offset], &d_c[i + offset], chunk_size * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }

    hipStreamSynchronize(stream0);
    hipStreamDestroy(stream0);
    hipStreamSynchronize(stream1);
    hipStreamDestroy(stream1);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    cout << "Testing vector scalar multiplication" << endl;
    cout << "Chunk size = " << chunk_size << endl;
    cout << "Time = " << elapsed_time << " ms" << endl;
    cout << "Acceleration = " << elapsed_time / chunk_size << endl
         << endl;

    if (chunk_size == 4)
    {
        fprintf(csv_file, "Chunk size; Elapsed time; Acceleration;\n");
    }
    fprintf(csv_file, "%d; %f; %f\n", chunk_size, elapsed_time, elapsed_time / chunk_size);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    return 0;
}