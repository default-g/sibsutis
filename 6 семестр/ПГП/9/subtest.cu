#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int *a, int *b, int *c, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int start = tid * size;
    int end = start + size;
    for (int i = start; i < end && i < size; i++)
    {
        c[tid * size + i] = a[tid * size + i] + b[tid * size + i];
    }
}

int main(int argc, char **argv)
{
    FILE *csv_file = fopen("results_9/2.csv", "a+");
    int size = pow(2, 16);
    int chunk_size = atoi(argv[1]);
    const int num_threads = 1024;
    int num_blocks = (size + num_threads - 1) / num_threads;
    int num_chunks = (size + chunk_size - 1) / chunk_size;
    int *h_a = new int[size];
    int *h_b = new int[size];
    int *h_c = new int[size];
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));
    srand(time(nullptr));
    for (int i = 0; i < size; i++)
    {
        h_a[i] = rand();
        h_b[i] = rand();
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

   
    for (int i = 0; i < num_chunks; i += 2)
    {
        int offset = num_chunks / 2;
        
        hipStream_t stream0, stream1;
        hipStreamCreate(&stream0);
        hipStreamCreate(&stream1);

        hipMemcpyAsync(&d_a[i], &h_a[i], chunk_size * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(&d_a[i + offset], &h_a[i + offset], chunk_size * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(&d_b[i], &h_b[i], chunk_size * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(&d_b[i + offset], &h_b[i + offset], chunk_size * sizeof(int), hipMemcpyHostToDevice, stream1);

        add<<<num_blocks, num_threads, 0, stream0>>>(&d_a[i], &d_b[i], &d_c[i], chunk_size);
        hipMemcpyAsync(&h_c[i], &d_c[i], chunk_size * sizeof(int), hipMemcpyDeviceToHost, stream0);
        add<<<num_blocks, num_threads, 0, stream1>>>(&d_a[i + offset], &d_b[i + offset], &d_c[i + offset], chunk_size);
        hipMemcpyAsync(&h_c[i + offset], &d_c[i + offset], chunk_size * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }

    hipStreamSynchronize(stream0);
    hipStreamDestroy(stream0);
    hipStreamSynchronize(stream1);
    hipStreamDestroy(stream1);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    cout << "Testing vector addition" << endl;
    cout << "Chunk size = " << chunk_size << endl;
    cout << "Number of chunks = " << num_chunks << endl;
    cout << "Time = " << elapsed_time << " ms" << endl;
    cout << "Acceleration = " << elapsed_time / chunk_size << endl
         << endl;

    if (chunk_size == 4)
    {
        fprintf(csv_file, "Chunk size; Elapsed time; Acceleration;\n");
    }
    fprintf(csv_file, "%d; %f; %f\n", chunk_size, elapsed_time, elapsed_time / chunk_size);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    return 0;
}
