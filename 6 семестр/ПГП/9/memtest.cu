#include <math.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

#define EPS 6

int main(int argc, char **argv)
{
    int size = atoi(argv[1]);
    float *h_data, *d_data;
    float *h_data_pinned;
    FILE *csv_file = fopen("results_9/1.csv", "a+");
    if (fgetc(csv_file) == EOF)
    {
        fprintf(csv_file, "Size; S: D → H; P: D → H; S: H → D; P: H → D;\n");
    }

    // Allocate memory on the host
    h_data = (float *)malloc(size * sizeof(float));
    hipHostMalloc((void **)&h_data_pinned, size * sizeof(float), hipHostMallocDefault);

    // Allocate memory on the device
    hipMalloc((void **)&d_data, size * sizeof(float));

    // Initialize data on the device
    hipMemset(d_data, 0, size * sizeof(float));
    hipMemcpy(d_data, h_data, size * sizeof(float), hipMemcpyHostToDevice);

    cout << "Testing memory copying (S - standart, P - pinned)" << endl;
    cout << "Size = " << size << endl;
    fprintf(csv_file, "%d; ", size);

    // Standard memory copy from device to host
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMemcpy(h_data, d_data, size * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "S: D → H = " << setprecision(EPS + 1) << elapsedTime << " ms" << endl;
    fprintf(csv_file, "%f; ", elapsedTime);

    // Pinned memory copy from device to host
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipEventRecord(start, 0);

    hipMemcpyAsync(h_data_pinned, d_data, size * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "P: D → H = " << setprecision(EPS) << elapsedTime << " ms" << endl;
    fprintf(csv_file, "%f; ", elapsedTime);

    // Standard memory copy from host to device
    hipEventRecord(start, 0);

    hipMemcpy(d_data, h_data, size * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "S: H → D = " << setprecision(EPS) << elapsedTime << " ms" << endl;
    fprintf(csv_file, "%f; ", elapsedTime);

    // Pinned memory copy from host to device
    hipEventRecord(start, 0);

    hipMemcpyAsync(d_data, h_data_pinned, size * sizeof(float), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "P: H → D = " << setprecision(EPS) << elapsedTime << " ms" << endl
         << endl;
    fprintf(csv_file, "%f\n", elapsedTime);

    // Free memory
    hipFree(d_data);
    free(h_data);
    hipHostFree(h_data_pinned);
    return 0;
}