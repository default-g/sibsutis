#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <thrust/gather.h>
#include <hipblas.h>

#define THREADS_PER_BLOCK 64

template <typename T>
void write_to_file(std::vector<std::vector<T>> data, std::string filename)
{
    std::ofstream out(filename);
    for (auto row : data)
    {
        std::string row_out = "";
        for (auto item : row)
        {
            row_out += std::to_string(item) + ";";
        }
        row_out.pop_back();
        out << row_out << std::endl;
    }
}

__global__ void simpleMatMulKernell(float *d_M, float *d_N, float *d_P, int N)
{
    int row = blockIdx.y * N + threadIdx.y;
    int col = blockIdx.x * N + threadIdx.x;
    if (row < N && col < N)
    {
        float product_val = 0;
        for (int k = 0; k < N; k++)
        {
            product_val += d_M[row * N + k] * d_N[k * N + col];
        }
        d_P[row * N + col] = product_val;
    }
}

__host__ std::vector<float> matrix_multiplication_compare(int N)
{
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    int matrix_size = N * N * sizeof(float);
    h_A = (float *)malloc(matrix_size);
    h_B = (float *)malloc(matrix_size);
    h_C = (float *)malloc(matrix_size);

    std::vector<float> stats{(float)N};

    hipMalloc(&d_A, matrix_size);
    hipMalloc(&d_B, matrix_size);
    hipMalloc(&d_C, matrix_size);

    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = rand() % 10;
        h_B[i] = rand() % 10;
    }

    hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    simpleMatMulKernell<<< (N / THREADS_PER_BLOCK - 1) + THREADS_PER_BLOCK, THREADS_PER_BLOCK>>> (d_A, d_B, d_C, N);
    hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    stats.push_back(milliseconds);


    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int lda = N, ldb = N, ldc = N;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    hipEventRecord(start);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc);
    hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);

    stats.push_back(milliseconds);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return stats;
}

int main()
{
    std::vector<std::vector<float>> statistics;
    for (int i = 2; i <= 1 << 13; i *= 2 ) {
        statistics.push_back(matrix_multiplication_compare(i));
    }

    write_to_file(statistics, "multiplication.csv");
        
}