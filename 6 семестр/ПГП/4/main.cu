
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 3
#define HEIGHT 4

__global__ void transpose1(int *in, int *out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WIDTH && y < HEIGHT) {
        out[x * HEIGHT + y] = in[y * WIDTH + x];
    }
}


__global__ void transpose2(int *in, int *out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WIDTH && y < HEIGHT) {
        out[x + y * WIDTH] = in[y + x * HEIGHT];
    }
}


int main() {
    int in[WIDTH * HEIGHT];
	for (int i = 0; i < WIDTH * HEIGHT; i++) {
		in[i] = i;
	}
    int out[HEIGHT * WIDTH];

    int *d_in, *d_out, *d2_out;

    hipMalloc((void **)&d_in, sizeof(int) * WIDTH * HEIGHT);
    hipMalloc((void **)&d_out, sizeof(int) * HEIGHT * WIDTH);
    hipMalloc((void **)&d2_out, sizeof(int) * HEIGHT * WIDTH);


    hipMemcpy(d_in, in, sizeof(int) * WIDTH * HEIGHT, hipMemcpyHostToDevice);

    dim3 grid((WIDTH + 15) / 16, (HEIGHT + 15) / 16);
    dim3 block(16, 16);

    transpose1<<<grid, block>>>(d_in, d_out);
    hipDeviceSynchronize();
    transpose2<<<grid, block>>>(d_in, d2_out);
    hipMemcpy(out, d_out, sizeof(int) * HEIGHT * WIDTH, hipMemcpyDeviceToHost);



    hipFree(d_in);
    hipFree(d_out);
    hipFree(d2_out);

    printf("Input matrix:\n");
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%d ", in[i * WIDTH + j]);
        }
        printf("\n");
    }

    printf("Output matrix:\n");
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < HEIGHT; j++) {
            printf("%d ", out[i * HEIGHT + j]);
        }
        printf("\n");
    }

    return 0;
}