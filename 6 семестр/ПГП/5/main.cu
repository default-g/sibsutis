
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 4
#define HEIGHT 4
#define BLOCK_SIZE 32


__global__ void transposeNaive(int *in, int *out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < HEIGHT && y < WIDTH) {
        out[x + y * HEIGHT] = in[y + x * WIDTH];
    }
}


__global__ void transposeNaiveWithSharedMemory(int *in, int *out) {
    __shared__ int cache[BLOCK_SIZE][BLOCK_SIZE];
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WIDTH && y < HEIGHT) {
        cache[threadIdx.y][threadIdx.x] = in[y * WIDTH + x];
    }
    __syncthreads();
    
    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;

     if (x < HEIGHT && y < WIDTH) {
        out[y * HEIGHT + x] = cache[threadIdx.x][threadIdx.y];
    }
}


__global__ void transposeWithUsingOfSharedMemoryAndBankConflictResolution(int *in, int *out) {
    __shared__ int tile[BLOCK_SIZE][BLOCK_SIZE + 1];
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    for (int j = 0; j < BLOCK_SIZE; j += blockDim.y)
    {
        if (x < WIDTH && y + j < HEIGHT)
        {
            int index_in = (y + j) * WIDTH + x;
            tile[threadIdx.y + j][threadIdx.x] = in[index_in];
        }
    }

    __syncthreads();

    x = blockIdx.y * BLOCK_SIZE + threadIdx.x;
    y = blockIdx.x * BLOCK_SIZE + threadIdx.y;
    for (int j = 0; j < BLOCK_SIZE; j += blockDim.y)
    {
        if (x < HEIGHT && y + j < WIDTH)
        {
            int index_out = (y + j) * HEIGHT + x;
            out[index_out] = tile[threadIdx.x][threadIdx.y + j];
        }
    }
     
}




int main() {
    int in[WIDTH * HEIGHT];
	for (int i = 0; i < WIDTH * HEIGHT; i++) {
		in[i] = i;
	}
    int out[HEIGHT * WIDTH];

    int *d_in, *d_out, *d2_out, *d3_out;

    hipMalloc((void **)&d_in, sizeof(int) * WIDTH * HEIGHT);
    hipMalloc((void **)&d_out, sizeof(int) * HEIGHT * WIDTH);
    hipMalloc((void **)&d2_out, sizeof(int) * HEIGHT * WIDTH);
    hipMalloc((void **)&d3_out, sizeof(int) * HEIGHT * WIDTH);

    hipMemcpy(d_in, in, sizeof(int) * WIDTH * HEIGHT, hipMemcpyHostToDevice);

    dim3 grid((WIDTH + 31) / 32, (HEIGHT + 31) / 32);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    hipDeviceSynchronize();
    transposeNaive<<<grid, block>>>(d_in, d_out);
    hipDeviceSynchronize();
    transposeNaiveWithSharedMemory<<<grid, block>>>(d_in, d2_out);
    hipDeviceSynchronize();
    transposeWithUsingOfSharedMemoryAndBankConflictResolution<<<grid, block>>>(d_in, d3_out);


    hipMemcpy(out, d3_out, sizeof(int) * HEIGHT * WIDTH, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d2_out);
    hipFree(d3_out);

    printf("Input matrix:\n");
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%d ", in[i * WIDTH + j]);
        }
        printf("\n");
    }

    printf("Output matrix:\n");
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < HEIGHT; j++) {
            printf("%d ", out[i * HEIGHT + j]);
        }
        printf("\n");
    }

    return 0;
}