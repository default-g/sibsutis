#include "hip/hip_runtime.h"

#include <chrono>

#include <math.h>
#include <stdio.h>

__global__ void sumOfVec(float *a, float *b, float *c) {
  c[threadIdx.x + blockDim.x * blockIdx.x] =
      a[threadIdx.x + blockDim.x * blockIdx.x] +
      b[threadIdx.x + blockDim.x * blockIdx.x];
}

int main() {
  float *ca, *a, *cb, *b, *cc, *c;
  FILE *fp = fopen("data_cuda.txt", "w");
  unsigned int N = pow(2, 20);
  for (int threads = 2; threads <= 1024; threads += 2) {
    unsigned int amount_of_blocks = N / threads;
    a = (float *)calloc(threads, sizeof(float));
    b = (float *)calloc(threads, sizeof(float));
    c = (float *)calloc(threads, sizeof(float));
    for (int j = 0; j < threads; j++) {
      a[j] = 1;
      b[j] = 1;
      c[j] = 1;
    }
    hipMalloc((void **)&ca, threads * sizeof(float));
    hipMalloc((void **)&cb, threads * sizeof(float));
    hipMalloc((void **)&cc, threads * sizeof(float));
    auto start = std::chrono::high_resolution_clock::now();
    sumOfVec<<<dim3(amount_of_blocks), dim3(threads)>>>(ca, cb, cc);
    hipMemcpy(a, ca, threads * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, cb, threads * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, cc, threads * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    free(a);
    free(b);
    free(c);
    hipFree(ca);
    hipFree(cb);
    hipFree(cc);
    std::chrono::duration<long double> difference = end - start;
    printf("Size of vector: %d, time: %Lf\n", threads, difference.count());
    fprintf(fp, "%Lf %d\n", difference.count(), threads);
  }

  return 0;
}