#include "hip/hip_runtime.h"

#include <chrono>

#include <math.h>
#include <stdio.h>

__global__ void sumOfVec(float *a, float *b, float *c) {
  c[threadIdx.x + blockDim.x * blockIdx.x] =
      a[threadIdx.x + blockDim.x * blockIdx.x] +
      b[threadIdx.x + blockDim.x * blockIdx.x];
}

int main() {
  float *ca, *a, *cb, *b, *cc, *c;
  FILE *fp = fopen("data_cuda.txt", "w");
  unsigned int N = 1 << 20;
   for(unsigned int threads = 1; threads <= 1024; threads = (threads == 1) ? 16 : threads * 2) {
    unsigned int amount_of_blocks = N / threads;
    a = (float *)calloc(N, sizeof(float));
    b = (float *)calloc(N, sizeof(float));
    c = (float *)calloc(N, sizeof(float));
    for (int j = 0; j < N; j++) {
      a[j] = 1;
      b[j] = 1;
      c[j] = 0;
    }
    hipMalloc((void **)&ca, N * sizeof(float));
    hipMalloc((void **)&cb, N * sizeof(float));
    hipMalloc((void **)&cc, N * sizeof(float));
    auto start = std::chrono::high_resolution_clock::now();
    sumOfVec<<<dim3(amount_of_blocks), dim3(threads)>>>(ca, cb, cc);
    hipMemcpy(a, ca, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, cb, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, cc, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    free(a);
    free(b);
    free(c);
    hipFree(ca);
    hipFree(cb);
    hipFree(cc);
    std::chrono::duration<long double> difference = end - start;
    printf("Size of vector: %d, time: %Lf\n", threads, difference.count());
    fprintf(fp, "%Lf %d\n", difference.count(), threads);
  }

  return 0;
}