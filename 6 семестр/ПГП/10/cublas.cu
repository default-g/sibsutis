#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void init_matrix(float *mat, int size)
{
    for (int i = 0; i < size * size; i++)
    {
        mat[i] = i;
    }
}

int main(int argc, char **argv)
{
    int N;
    N = atoi(argv[1]);
    if (argc < 2)
    {
        printf("INVALID INPUT!\n");
        exit(EXIT_FAILURE);
    }

    FILE *csv_file = fopen("cublas_res.csv", "a+");

    float *A, *B, *C;
    float alpha = 1.0, beta = 0.0;
    int size = N * N;

    A = (float *)malloc(size * sizeof(float));
    B = (float *)malloc(size * sizeof(float));
    C = (float *)malloc(size * sizeof(float));

    init_matrix(A, N);
    init_matrix(B, N);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size * sizeof(float));
    hipMalloc(&d_B, size * sizeof(float));
    hipMalloc(&d_C, size * sizeof(float));

    hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta, d_C, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("cuBLAS: %d^2 / %f ms\n", N, elapsed_time);
    fprintf(csv_file, "%d; %f;\n", size, elapsed_time);

    hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(handle);

    return 0;
}
